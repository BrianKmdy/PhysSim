#include "hip/hip_runtime.h"
#include <algorithm>

#include "Types.h"
#include "Simulate.cuh"

std::vector<Particle*> gDeviceParticles;
std::vector<Box*> gDeviceBoxes;

__host__ void initializeCuda(Instance* instance)
{
	int nDevices;
	hipGetDeviceCount(&nDevices);

	for (int i = 0; i < nDevices; i++) {
		hipSetDevice(i);

		gDeviceParticles.push_back(nullptr);
		hipMalloc(&gDeviceParticles[i], instance->nParticles * sizeof(Particle));
		gDeviceBoxes.push_back(nullptr);
		hipMalloc(&gDeviceBoxes[i], instance->nBoxes * sizeof(Box));
	}
}

__host__ void unInitializeCuda()
{
	int nDevices;
	hipGetDeviceCount(&nDevices);

	for (int i = 0; i < nDevices; i++) {
		hipSetDevice(i);

		hipFree(gDeviceParticles[i]);
		hipFree(gDeviceBoxes[i]);
		hipDeviceReset();
	}
}

__host__ std::chrono::milliseconds simulate(Instance* instance, Particle* particles, Box* boxes, int kernel)
{
	for (int i = 0; i < instance->nBoxes; i++) {
		boxes[i].mass = 0.0;
		boxes[i].centerMass = make_float2(0.0, 0.0);
		boxes[i].nParticles = 0;
		boxes[i].particleOffset = 0;
	}

	for (int i = 0; i < instance->nParticles; i++) {
		particles[i].force = make_float2(0.0, 0.0);
		particles[i].boxId = instance->getBoxIndex(particles[i].position);
		boxes[particles[i].boxId].centerMass = (boxes[particles[i].boxId].centerMass * boxes[particles[i].boxId].mass + particles[i].position * particles[i].mass)
			/ (boxes[particles[i].boxId].mass + particles[i].mass);
		boxes[particles[i].boxId].mass += particles[i].mass;
		boxes[particles[i].boxId].nParticles += 1;
	}

	// Is it possible to do this in a more efficient way without using a sort?
	std::sort(particles, particles + instance->nParticles,
		[](const Particle& a, const Particle& b) {
			return a.boxId < b.boxId;
	});

	int boxId = -1;
	for (int i = 0; i < instance->nParticles; i++) {
		if (particles[i].boxId != boxId) {
			boxes[particles[i].boxId].particleOffset = i;
			boxId = particles[i].boxId;
		}
	}

	int nDevices;
	hipGetDeviceCount(&nDevices);

	// Copy the instance to device memory and run the kernel
	auto kernelStartTime = getMilliseconds();
	int blockSize = (instance->nParticles + nThreads - 1) / nThreads;
	int deviceBatchSize = (instance->nParticles + nDevices - 1) / nDevices;
	for (int i = 0; i < nDevices; i++) {
		hipSetDevice(i);

		int endIndex = static_cast<int>(std::min((i + 1) * deviceBatchSize, instance->nParticles));
		gpuErrchk(hipMemcpy(gDeviceParticles[i], particles, instance->nParticles * sizeof(Particle), hipMemcpyHostToDevice));
		gpuErrchk(hipMemcpy(gDeviceBoxes[i], boxes, instance->nBoxes * sizeof(Box), hipMemcpyHostToDevice));

		// Launch the kernel
		switch (kernel) {
			case Kernel::experimental:
				experimental<<<blockSize, nThreads>>>(i, deviceBatchSize, endIndex, *instance, gDeviceParticles[i], gDeviceBoxes[i]);
				break;
			default:
				gravity<<<blockSize, nThreads>>>(i, deviceBatchSize, endIndex, *instance, gDeviceParticles[i], gDeviceBoxes[i]);
				break;
		}

		gpuErrchk(hipPeekAtLastError());
	}

	// Synchronize with devices and copy the udpated instance back
	for (int i = 0; i < nDevices; i++) {
		hipSetDevice(i);

		int numElements = std::min(deviceBatchSize, instance->nParticles - (i * deviceBatchSize));
		gpuErrchk(hipDeviceSynchronize());
		gpuErrchk(hipMemcpy(particles + (i * deviceBatchSize), gDeviceParticles[i] + (i* deviceBatchSize), numElements * sizeof(Particle), hipMemcpyDeviceToHost));
	}
	auto kernelEndTime = getMilliseconds();

	// Calculate each particle's new position and velocity based on its force for this frame
	for (int i = 0; i < instance->nParticles; i++) {
		float2 acceleration = particles[i].force / particles[i].mass;

		particles[i].position += (particles[i].velocity * instance->timeStep) + (0.5 * acceleration * powf(instance->timeStep, 2.0));
		particles[i].velocity += acceleration * instance->timeStep;
		particles[i].enforceBoundary(instance->maxBoundary);
	}

	return kernelEndTime - kernelStartTime;
}

__global__ void gravity(int deviceId, int deviceBatchSize, int endIndex, Instance instance, Particle* particles, Box* boxes)
{
	unsigned int index = deviceId * deviceBatchSize + blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int stride = blockDim.x * gridDim.x;

	for (int i = index; i < endIndex; i += stride) {
	 	for (int o = 0; o < instance.nBoxes; o++) {
			if (o == particles[i].boxId) {
				for (int p = boxes[o].particleOffset; p < boxes[o].particleOffset + boxes[o].nParticles; p++) {
					// XXX/bmoody Can review making this more efficient, is it necessary to square/sqrt dist so much?
					float dist = distance(particles[i].position, particles[p].position);
					if (dist > instance.minForceDistance)
						particles[i].force += (direction(particles[i].position, particles[p].position) / dist) * ((particles[i].mass * particles[p].mass) / powf(dist, 2.0));
				}
			}
			else
			{
				float dist = distance(particles[i].position, boxes[o].centerMass);
				if (dist > instance.minForceDistance)
					particles[i].force += (direction(particles[i].position, boxes[o].centerMass) / dist) * ((particles[i].mass * boxes[o].mass) / powf(dist, 2.0));
			}
	 	}
	}
}

// XXX/bmoody Can make this more accurate (expensive) by having each particle directly interact with the particles of adjacent boxes as well
__global__ void experimental(int deviceId, int deviceBatchSize, int endIndex, Instance instance, Particle* particles, Box* boxes)
{
	unsigned int index = deviceId * deviceBatchSize + blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int stride = blockDim.x * gridDim.x;

	for (int i = index; i < endIndex; i += stride) {
		for (int o = 0; o < instance.nBoxes; o++) {
			if (o == particles[i].boxId) {
				for (int p = boxes[o].particleOffset; p < boxes[o].particleOffset + boxes[o].nParticles; p++) {
					// XXX/bmoody Can review making this more efficient, is it necessary to square/sqrt dist so much?
					float dist = distance(particles[i].position, particles[p].position);
					if (dist > instance.minForceDistance)
						particles[i].force += (direction(particles[p].position, particles[i].position) / dist) * ((particles[i].mass * particles[p].mass) / powf(dist, 2.0));
				}
			}
			else
			{
				float dist = distance(particles[i].position, boxes[o].centerMass);
				if (dist > instance.minForceDistance)
					particles[i].force += (direction(boxes[o].centerMass, particles[i].position) / dist) * ((particles[i].mass * boxes[o].mass) / powf(dist, 2.0));
			}
		}
	}
}

__host__ __device__ int Instance::getBoxIndex(float2 position)
{
	int2 index = (position + (dimensions / 2)) / boxSize;

	return index.x * divisions + index.y;
}

__host__ __device__ void Particle::enforceBoundary(float maxBoundary)
{
	//// XXX/bmoody Review this, there must be a better way
	if (position.x < -maxBoundary) {
		position.x = -maxBoundary + 1.0;
		velocity.x = 0.0;
	}
	if (position.x > maxBoundary) {
		position.x = maxBoundary - 1.0;
		velocity.x = 0.0;
	}
	if (position.y < -maxBoundary) {
		position.y = -maxBoundary + 1.0;
		velocity.y = 0.0;
	}
	if (position.y > maxBoundary) {
		position.y = maxBoundary - 1.0;
		velocity.y = 0.0;
	}
}